
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

const int TRY_COUNT = 512;

void fetch_data(char *file_name, float **mat1, float **mat2, int *mat_size) {
    FILE *file = fopen(file_name, "r");

    fscanf(file, "SizeA= %d", mat_size);
    float *matA = (float*) malloc(sizeof(float) * (*mat_size));
    for (int i = 0; i < (*mat_size); i++) {
        fscanf(file, "%f", &(matA[i]));
    }

    fscanf(file, " ");
    
    fscanf(file, "SizeB= %d", mat_size);
    float *matB = (float*) malloc(sizeof(float) * (*mat_size));
    for (int i = 0; i < (*mat_size); i++) {
        fscanf(file, "%f", &(matB[i]));
    }

    *mat1 = matA;
    *mat2 = matB;
    fclose(file);
}

__global__
void dot_product(float *mat1, float *mat2, float *tmp, int *mat_size, float *result) {
    tmp[threadIdx.x] = mat1[threadIdx.x] * mat2[threadIdx.x];
    __syncthreads();

    if(threadIdx.x == 0) {
        float sum = 0;
        for(int i = 0; i < (*mat_size); i++) {
            sum += tmp[i];
        }
        *result = sum;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Please provide file name!");
        exit(1);
    }

    float *mat1, *mat2;
    int mat_size;
    fetch_data(argv[1], &mat1, &mat2, &mat_size);
    
    float *cuda_mat1, *cuda_mat2, *cuda_tmp, *cuda_result;
    int *cuda_mat_size;
    hipMalloc(&cuda_mat1, sizeof(float) * mat_size);
    hipMalloc(&cuda_mat2, sizeof(float) * mat_size);
    hipMalloc(&cuda_tmp, sizeof(float) * mat_size);
    hipMalloc(&cuda_mat_size, sizeof(int));
    hipMalloc(&cuda_result, sizeof(float));
    hipMemcpy(cuda_mat1, mat1, sizeof(float) * mat_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_mat2, mat2, sizeof(float) * mat_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_mat_size, &mat_size, sizeof(int), hipMemcpyHostToDevice);

    double time_total = 0;
    for (int i = 0; i < TRY_COUNT; i++) {
        double start = omp_get_wtime();
        dot_product<<< 1, mat_size >>>(cuda_mat1, cuda_mat2, cuda_tmp, cuda_mat_size, cuda_result);
        hipDeviceSynchronize();
        double end = omp_get_wtime();
        time_total += end - start;
    }

    float result;
    hipMemcpy(&result, cuda_result, sizeof(float), hipMemcpyDeviceToHost);
    printf("Result: %f\n", result);
    printf("Time: %fus\n", 1000000 * (time_total / ((double) TRY_COUNT)));
    hipFree(cuda_mat1);
    hipFree(cuda_mat2);
    hipFree(cuda_tmp);
    hipFree(cuda_mat_size);
    hipFree(cuda_result);
    return 0;
}
